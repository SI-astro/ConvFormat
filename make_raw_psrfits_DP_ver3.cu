#include "hip/hip_runtime.h"
/*

This is a simple dedispersion code using CUDA.
double precision ver

Note : some values for de-dispersion was copied from Mikami-san's code

S.Ikebe,
*/

#include "make_raw_psrfits_kuma.h"
#include "utils/utils_si.cuh"



int getFileSize(const char* fileName)
{
  struct stat st_f;

  if (stat(fileName, &st_f) != 0) {
    return -1;
  }

  //check file type
  if ((st_f.st_mode & S_IFMT) != S_IFREG) {
    //S_IFMT : type
    //S_IFREG : regular file
    return -1;
  }

  return st_f.st_size;//file size in bite
}


__global__
void bitconversion(unsigned char *d_raw_pre, hipfftDoubleReal *d_real, unsigned char *wordl, unsigned char *wordr, unsigned char *alword)
  {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if( i < N_dt/4){
     wordl[i] = (d_raw_pre[i] & 0xAA) >> 1;
     wordr[i] = (d_raw_pre[i] & 0x55) << 1;
     alword[i] = (wordl[i] | wordr[i]) ^ 0x33;
     d_real[i*4] = alword[i] & 0x03;
     d_real[i*4+1] = (alword[i] >> 2) & 0x03;
     d_real[i*4+2] = (alword[i] >> 4) & 0x03;
     d_real[i*4+3] = (alword[i] >> 6) & 0x03;
      }//for j
    }//if

__global__
void bitconversion_i(unsigned char *d_raw_pre, hipfftDoubleReal *d_real, hipfftDoubleReal *ho)
  {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int k;
    k = d_raw_pre[i];
    d_real[4*i] = ho[4*k];
    d_real[4*i+1] = ho[4*k+1];
    d_real[4*i+2] = ho[4*k+2];
    d_real[4*i+3] = ho[4*k+3];
    /*
    if (d_real[i]!=0){
    printf("d_real: %lf",d_real[i]);}*/
    //printf("d_real: %lf",ho[i]);
    }//if

__global__
void bitconversion_k(unsigned char *d_raw_pre, hipfftDoubleReal *d_real)
  {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    d_real[i] = d_raw_pre[i];
    }//if


int read_Kashima_to_d(unsigned char *raw_pre, hipfftDoubleReal *d_real, FILE **fp, int length){
      dim3 block_bit (SET_BLOCK, 1, 1);
      dim3 grid_bit  (length / block_bit.x, 1, 1);
      long long int err_read; hipError_t err;
      double time_s[4];
      double time_e[4];

      time_s[0] = cputimeinsec();
      err_read = fread(raw_pre,sizeof(unsigned char),length,*fp);
      if(err_read!=length){
       printf("data is shorter than N_dt so, stop this roop...\n");
       return 1;
      }
      unsigned char *d_raw_pre;
      err = hipMalloc((void**)&d_raw_pre, sizeof(unsigned char)*length);
       if (err != hipSuccess) {
       exit(err);
       }
      hipDeviceSynchronize();
      time_e[0] = cputimeinsec();

      time_s[1] = cputimeinsec();
      hipMemcpy(d_raw_pre, raw_pre, length*sizeof(unsigned char), hipMemcpyHostToDevice);
      hipDeviceSynchronize();
      time_e[1] = cputimeinsec();

      time_s[2] = cputimeinsec();
      bitconversion_k<<<grid_bit,block_bit>>>(d_raw_pre, d_real);
      hipDeviceSynchronize();
      time_e[2] = cputimeinsec();

      time_s[3] = cputimeinsec();
      hipFree(d_raw_pre);
      hipDeviceSynchronize();
      time_e[3] = cputimeinsec();

          /*
          printf("time for fread + malloc for bit conversion : %lf\n", time_e[0]-time_s[0]);
          printf("time for memcpy of raw data : %lf\n", time_e[1]-time_s[1]);
          printf("time for bit conversion : %lf\n", time_e[2]-time_s[2]);
          printf("time for free : %lf\n", time_e[3]-time_s[3]);
          */
      return 0;
    }



int read_Ibaraki_Y_to_d(unsigned char *raw_pre, double *d_real, FILE **fp, hipfftDoubleReal *ho, double *time_arranging_bit, double *time_fread){
      unsigned char  *d_raw_pre; //*raw_pre
      dim3 block_bit (SET_BLOCK, 1, 1);
      dim3 grid_bit  (N_dt / block_bit.x / 4, 1, 1);
      long long int err_read;
      hipError_t err;
      double time_s[5];
      double time_e[5];

      //start for Hitachi or Takahagi data
      printf("data is read as observation in Hitachi or Takahagi\n");
      time_s[0] = cputimeinsec();
      err = hipMalloc((void**)&d_raw_pre, sizeof(unsigned char)*N_dt*2/8);
      if (err != hipSuccess) {
        exit(err);
      }
      hipDeviceSynchronize();
      time_e[0] = cputimeinsec();

      time_s[1] = cputimeinsec();
      err_read = fread(raw_pre,sizeof(unsigned char),(N_dt)*2/8,*fp);
      if(err_read!=(long long int)(N_dt)*INPUT_BITSIZE/8){
        printf("data is shorter than N_dt so, stop this roop...\n");
        //break;
        return 1;
        exit(0);
      }
      time_e[1] = cputimeinsec();

      time_s[2] = cputimeinsec();
      hipMemcpy(d_raw_pre, raw_pre, N_dt*sizeof(unsigned char)*2/8, hipMemcpyHostToDevice);
      hipDeviceSynchronize();
      time_e[2] = cputimeinsec();

      time_s[3] = cputimeinsec();
      bitconversion_i<<<grid_bit,block_bit>>>(d_raw_pre, d_real,ho);
      hipDeviceSynchronize();
      time_e[3] = cputimeinsec();

      time_s[4] = cputimeinsec();
      hipFree(d_raw_pre); //free(raw_pre);
      hipDeviceSynchronize();
      time_e[4] = cputimeinsec();

      //end for Hitachi or Takahagi data
      printf("time for malloc for bit conversion : %lf\n", time_e[0]-time_s[0]);
      printf("time for fread : %lf\n", time_e[1]-time_s[1]);
      printf("time for memcpy of raw data : %lf\n", time_e[2]-time_s[2]);
      printf("time for bit conversion : %lf\n", time_e[3]-time_s[3]);
      printf("time for free : %lf\n", time_e[4]-time_s[4]);

      *time_arranging_bit += (time_e[3] - time_s[3]);
      *time_fread += (time_e[1]-time_s[1]);

      return 0;
    }

int read_Kashima(double *h_in, FILE **fp, int length){
  unsigned char *raw;
  int err_read;
  raw = (unsigned char*) malloc(sizeof(unsigned char) * length);
  if(raw == NULL) {
    printf("memory cannot be allocated!!\n");
    exit(EXIT_FAILURE);
  }
  err_read = fread(raw,sizeof(unsigned char),length,*fp);
  if(err_read!=length){
    printf("data is shorter than N_dt so, stop this roop...\n");
    //break;
    return 1;
  }
  for(int i=0;i<length;i++){
    h_in[i] = raw[i];
  }
  free(raw);
  return 0;
}


__global__ void sum_time_gpu0(float *time_gpu_dev, float milliseconds){
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx == 0){
  time_gpu_dev[idx] = time_gpu_dev[idx] + milliseconds/1e3;
  }
}

__global__ void sum_time_gpu1(float *time_gpu_dev, float milliseconds){
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx == 1){
  time_gpu_dev[idx] = time_gpu_dev[idx] + milliseconds/1e3;
  }
}

__global__ void sum_time_gpu2(float *time_gpu_dev, float milliseconds){
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx == 2){
  time_gpu_dev[idx] = time_gpu_dev[idx] + milliseconds/1e3;
  }
}


//#ZZZ
__global__ void realtocomplex(hipfftDoubleReal *in, hipfftDoubleComplex *out, int bin_size){
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < bin_size){
    out[idx].x   = in[idx];
    out[idx].y   = 0;//in[idx];
    //out[idx][1]   = 0;
  }
}
__global__ void realtocomplex_single(hipfftReal *in, hipfftComplex *out, int bin_size){
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < bin_size){
    out[idx].x   = in[idx];
    out[idx].y   = 0;//in[idx];
    //out[idx][1]   = 0;
  }
}





__global__ void reduce4(double *g_idata, double *g_odata, unsigned int n)
{
    extern __shared__ double sdata[];

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    double mysum = (i < n) ? g_idata[i] : 0;
    if (i + blockDim.x < n) mysum += g_idata[i + blockDim.x];
    sdata[tid] = mysum;
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
        if (tid < s) {
            sdata[tid] = mysum = mysum + sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid < 32) {
       if(blockDim.x >= 64) mysum += sdata[tid + 32];
        for (int offset = 32/2; offset>0; offset>>=1) {
            mysum += __shfl_down(mysum, offset);
        }
    }
    if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}

__global__
void cp_array(double *arr_dev, double *d_out, int repeat, int intgnumber){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i >= 0 && i < intgnumber){
  arr_dev[i] = d_out[repeat*intgnumber+i];
  }
}

__global__
void input_result(double *res_dev, double *out, int repeat){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i==repeat){
  	res_dev[i] = out[0];
  }
}

__global__
void sum_to_ave(double *ave_dev, double *res_dev, int intgnumber, int max_repeat){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i >= 0 && i < max_repeat){
        res_dev[i] = ave_dev[i] / intgnumber / N_dt;
  }
}


void exec_reduce4(double *d_real_out, double *res_dev, int intgnumber, int timedelay, int max_repeat){
  double *out1_dev, *out2_dev;
  int th = SET_BLOCK;
  int blocks = (intgnumber - 1) / (2 * th) + 1;
  int shared_mem_size = 2 * th * sizeof(double);
  hipMalloc((void**)&out1_dev, sizeof(double) * blocks);
  hipMalloc((void**)&out2_dev, sizeof(double) * blocks);
  int repeat;
  double *arr_dev, *ave_dev;
  hipMalloc((void**)&arr_dev, sizeof(double)*intgnumber);
  hipMalloc((void**)&ave_dev, sizeof(double)*max_repeat);
  //hipMalloc((void**)&res_dev, sizeof(double)*max_repeat);
  int blocksize_cp = SET_BLOCK;//512;
  dim3 block_cp (blocksize_cp, 1, 1);
  dim3 grid_cp  (N_dt / block_cp.x, 1, 1);//here +1 is reqired as FFTed array has (N_dt/2+1) samples
  dim3 block_in (SET_BLOCK, 1, 1);
  dim3 grid_in (max_repeat / block_in.x+1, 1, 1);

  for(repeat = 0; repeat < max_repeat; repeat++){
  cp_array<<<grid_cp, block_cp>>>(arr_dev, d_real_out, repeat, intgnumber);
  double **in = &arr_dev, **out = &out1_dev;
  int n = intgnumber;
  //printf("%")
  while (blocks > 1) {
    reduce4<<<blocks, th, shared_mem_size>>>(*in, *out, intgnumber);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        puts(hipGetErrorString(err));
    }
    if (*out == out1_dev) {
        out = &out2_dev; in = &out1_dev;
    }
    else {
        out = &out1_dev; in = &out2_dev;
    }
    n = blocks;
    blocks = (blocks - 1) / (2 * th) + 1;
    hipDeviceSynchronize();
    }
    reduce4<<<blocks, th, shared_mem_size>>>(*in, *out, n);

    //dim3 block_in (SET_BLOCK, 1, 1);
    //dim3 grid_in (max_repeat / block_in.x+1, 1, 1);
    input_result<<<grid_in, block_in >>>(ave_dev, *out, repeat);
    //res_dev[repeat] = out;
    //hipMemcpy(&result, *out, sizeof(double), hipMemcpyDeviceToHost);
    blocks = (intgnumber - 1) / (2 * th) + 1;
  }
  hipFree(out1_dev); hipFree(out2_dev); hipFree(arr_dev);
  sum_to_ave<<<grid_in, block_in>>>(ave_dev, res_dev, intgnumber, max_repeat);
  hipFree(ave_dev);
}

__global__ void reduceUnrollWarps1 (double *g_idata, double *g_odata)
{
    int tid = threadIdx.x;
    //long idx = blockIdx.x * blockDim.x * 1 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    double *idata = g_idata + blockIdx.x * blockDim.x * 1;

    // unrolling 8
    /*
    if (idx + 1 * blockDim.x < n)
    {
        double a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1; // + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    */

    __syncthreads();

    // in-place reduction and complete unroll
    /*
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();
    */

    // unrolling warp
    if (tid < 32)
    {
        volatile double *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
    //if (tid == 0) printf("summed %lf",g_odata[blockIdx.x]);
}

void sum64(double *d_real_2d, double *d_real_2dsummed){
  dim3 grid (64, 1, 1);
  dim3 block  (FFTP/ 2 / grid.x, 1, 1);
  reduceUnrollWarps1<<<block,grid>>>(d_real_2d, d_real_2dsummed);
}

//--------end of this code
