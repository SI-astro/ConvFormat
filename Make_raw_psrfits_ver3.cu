#include "hip/hip_runtime.h"
/*
This code makes psrfits data from the data observed by Japanese telescpoe.
*/

#include "make_raw_psrfits_kuma.h"
#include "utils/utils_si.cuh"

__global__
void make_ho(hipfftDoubleReal *ho){
  unsigned char i = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned char wordl, wordr, alword;
  wordl = (i & 0xAA) >> 1;
  wordr = (i & 0x55) << 1;
  alword = wordl | wordr;
  alword = alword ^ 0x33;
  ho[4*i] = alword & 0x03;
  ho[4*i+1] = (alword >> 2) & 0x03;
  ho[4*i+2] = (alword >> 4) & 0x03;
  ho[4*i+3] = (alword >> 6) & 0x03;
  //printf("d_ho: %lf",ho[i]);
}

__global__
void calcmag_to_double(hipfftDoubleComplex *data, double *mag, int bin_size){
      int idx = threadIdx.x + blockDim.x*blockIdx.x;
      mag[idx]   = data[idx].x * data[idx].x + data[idx].y * data[idx].y;
      //if (isnan(mag[idx])){printf("nan in calcmag");}
    }

__global__
void cp_array_cufftDoubleReal(hipfftDoubleReal *in, hipfftDoubleReal *out, int in_start, int out_start){
      int i = blockIdx.x*blockDim.x + threadIdx.x;
      out[out_start+i] = in[in_start+i];
    }

__global__
void cp_array_double(double *in, double *out, int in_start, int out_start){
          int i = blockIdx.x*blockDim.x + threadIdx.x;
          out[out_start+i] = in[in_start+i];
    }

    __global__
    void double_devide(double *in, long length){ //割りたい配列と割りたい値
      int i = blockIdx.x*blockDim.x + threadIdx.x;
      in[i] = in[i]/length;
    }

int main(){

  hipError_t err;

  //for time measure
  double t_start,t_end;
  t_start = cputimeinsec();
  clock_t start_clock_all = clock();
  time_t start_time_all, end_time_all;//for CPU time
  start_time_all = time(NULL);
  //for cpu
  int time_read, time_ave, time_out;
  time_read = 0; time_ave = 0; time_out = 0;
  time_t time_read_s, time_read_e, time_ave_s, time_ave_e, time_out_s, time_out_e;
  //for hipMemcpy
  double time_s[1], time_e[1], time_res[1];
  time_res[0]=0.0;
  double time_arranging_bit = 0; double time_fread = 0;
  float time_arrange_bit_bycudaE = 0.0;
  float h_time_memcpy = 0.0; float h_time_memcpy_a = 0.0;

  //file pointer
  FILE *fp,*fp_out;
  printf("\n----------------------------\n");

  const char *readname = INPUT_RAWDATA;
  //sprintf(filename,"%s",inputname);
  if ((fp = fopen(readname, "rb")) == NULL){
    //  if ((fp = fopen("CRAB_2009222233800-233802.raw", "rb")) == NULL){
    printf("file open error!!\n");
    exit(EXIT_FAILURE);
  }
  printf("read data from %s \n" , INPUT_RAWDATA);

  long n_move;

  const char *writename = OUTPUT_DATA;
  if ((fp_out = fopen(writename, "wb")) == NULL){
          printf("file open error!!\n");
          exit(EXIT_FAILURE);
        }


  //size of input data
  fseek(fp,0,SEEK_END);
  long long length = ftell(fp);
  printf("Length in byte  = %lld\n", length);
  int totRp = length * 8/INPUT_BITSIZE / SR;
  printf("Observation period: %d seconds\n", totRp);
  rewind(fp);
  int loop_max = length * 8/INPUT_BITSIZE / N_dt;

  int fft_max = N_dt / FFTP;
  double dt = double(FFTP)/SR;
  printf("dt %lf\n",dt);

  dim3 block (SET_BLOCK, 1, 1); dim3 grid  (FFTP /2 / block.x, 1, 1);
  dim3 grid_ini  (FFTP / block.x, 1, 1);

  hipfftHandle plan_f;
  hipfftPlan1d(&plan_f, FFTP, HIPFFT_D2Z, 1);

  unsigned char *raw_pre;
  raw_pre = (unsigned char*) malloc(sizeof(unsigned char) * N_dt * INPUT_BITSIZE/8);
  if(raw_pre == NULL) {
    printf("memory cannot be allocated!!\n");
    exit(EXIT_FAILURE);
  }
  double *res_host, *freq;
    res_host = (double*) malloc(sizeof(double) * FFTP /2/64);
    if(res_host == NULL) {
      printf("memory cannot be allocated in res_host!!\n");
      exit(EXIT_FAILURE);
    }
    freq = (double*) malloc(sizeof(double) * FFTP /2/64);
    if(freq == NULL) {
      printf("memory cannot be allocated in res_host!!\n");
      exit(EXIT_FAILURE);
    }
  /*
  for(int i=0; i<FFTP /2/64;i++){
    freq[i] = FREQ_DATA_LOW + (FREQ_DATA_WIDTH/(FFTP /2/64))*i; //MHz
  }
  */
  for(int i=0; i<FFTP /2/64;i++){
    freq[i] = FREQ_DATA_LOW + FREQ_DATA_WIDTH - (FREQ_DATA_WIDTH/(FFTP /2/64))*i; //MHz
  }
  double *d_real_out, *d_real_2d, *d_real_2dsummed, *d_real_out_sum;
    err = hipMalloc((void**)&d_real_2d, sizeof(double)*N_dt/2);
      if (err != hipSuccess) {printf("error in cudamalloc in d_real_2d\n"); exit(err);}
    err = hipMalloc((void**)&d_real_out, sizeof(double)*FFTP/2);
      if (err != hipSuccess) {
        printf("error in cudamalloc in d_real_out\n");
        exit(err);
      }
      err = hipMalloc((void**)&d_real_out_sum, sizeof(double)*FFTP/2/64);
        if (err != hipSuccess) {
          printf("error in cudamalloc in d_real_sum_out\n");
          exit(err);
        }
    err = hipMalloc((void**)&d_real_2dsummed, sizeof(double)*N_dt/2/64);
      if (err != hipSuccess) {
        printf("error in cudamalloc in d_real_2dsummed\n");
        exit(err);
    }

  hipfftDoubleReal *d_real, *d_real_ini;
  hipfftDoubleComplex *d_cplx;
    err = hipMalloc((void**)&d_real, sizeof(hipfftDoubleReal)*N_dt);
      if (err != hipSuccess) {printf("error in cudamalloc in d_real\n"); exit(err);}
    err = hipMalloc((void**)&d_cplx, sizeof(hipfftDoubleComplex)*FFTP);
    if (err != hipSuccess) {
      printf("error in cudamalloc in d_cplx\n");
      exit(err);
    }
    err = hipMalloc((void**)&d_real_ini, sizeof(hipfftDoubleReal)*FFTP);
    if (err != hipSuccess) {
      printf("error in cudamalloc in d_real_ini\n");
      exit(err);
    }
  hipfftDoubleReal *ho;
  hipMalloc((void**)&ho, sizeof(hipfftDoubleReal)*1024);
  make_ho<<<1,256>>>(ho);

  //roop for larger data
  for(int tt=0;tt<loop_max;tt++){
    if(tt==0){n_move = long(sizeof(unsigned char)*N_dt*tt*INPUT_BITSIZE/8);}
    //else{n_move = long(sizeof(unsigned char)*N_dt*tt-i_d_tp);}
    else{n_move = long(sizeof(unsigned char)*(N_dt)*tt*INPUT_BITSIZE/8);}
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    printf("roop of %d %ld\n",tt,n_move);
    fseek(fp, n_move, SEEK_SET); //move position of pointer by n_move byte from start of the file

    int e;
    time_read_s = time(NULL);
      if(observationdata == "Ibaraki"){
      //start for Hitachi or Takahagi data
      e = read_Ibaraki_Y_to_d(raw_pre, d_real, &fp, ho, &time_arranging_bit,&time_fread);
      }
      else if(observationdata == "Kashima"){
      e = read_Kashima_to_d(raw_pre, d_real, &fp, N_dt);
      }
      else{
      printf("!!!error in reading data!!!!\n");
      break;
      }
      if(!(e == 0)){
      printf("data is shorter than N_dt so, stop this roop...\n");
      exit(0);
      break;
      }
      time_read_e = time(NULL);
      time_read += (time_read_e - time_read_s);


    //-----------------start
    time_s[0] = cputimeinsec();
    for(int i=0;i<fft_max;i++){ //i<8000
      cp_array_cufftDoubleReal<<<grid_ini, block>>>(d_real, d_real_ini, i*FFTP, 0);
      hipfftExecD2Z(plan_f, d_real_ini, d_cplx);
      calcmag_to_double<<<grid, block>>>(d_cplx, d_real_out, FFTP/2);
      //cp_array_double<<<grid, block>>>(d_real_out, d_real_2d, 0, i*FFTP/2);
      time_ave_s = time(NULL);
      sum64(d_real_out,d_real_out_sum);
      time_ave_e = time(NULL);
      time_ave += (time_ave_e - time_ave_s);

      hipMemcpy(res_host, d_real_out_sum, sizeof(double)*(FFTP/128), hipMemcpyDeviceToHost);
      //printf("%lf",res_host[0]);
      time_out_s = time(NULL);
      
      for (long j=0;j<FFTP/128;j++){
     // fwrite(res_host, sizeof(double), FFTP/128, fp_out); // -> this seems to work well
      //fprintf(fp_out, "%hhu ", (unsigned char)(res_host[FFTP/128-j+1]));
      //fprintf(fp_out, "%f ", (float)(res_host[FFTP/128-j+1]));

      //fprintf(fp_out, "%e\t%lf\t%f\n", i*dt+tt*((double)N_dt/SR),freq[j],(float)(res_host[FFTP/128-j-1])); //for test
      fprintf(fp_out, "%f\n", (float)(res_host[FFTP/128-j-1]));

      //fprintf(fp_out, "%e\t%lf\t%hhu\n", i*dt+tt*((double)N_dt/SR),freq[j],(unsigned char)(res_host[j])); // unsigned char seems bad
      //fprintf(fp_out, "%e\t%lf\t%lf\n", i*dt+tt*((double)N_dt/SR),freq[j],res_host[j]);
      }

    }
    time_e[0] = cputimeinsec();
    time_res[0] += (time_e[0] - time_s[0]);


    }//tt
    hipFree(d_real_out); free(res_host); free(freq); hipFree(ho); hipFree(d_real_out_sum);
    hipfftDestroy(plan_f);
    fclose(fp); fclose(fp_out);


  clock_t end_clock_fwrite = clock();
  //printf("fwrite clock_t %2.5f sec\n",(double)(end_clock_fwrite - start_clock_fwrite) / CLOCKS_PER_SEC);
  clock_t end_clock_all = clock();
  end_time_all = time(NULL);


  printf("\n----------------------------\n");
  printf("summary\n");
  printf("in total %2.5f sec\n",(double)(end_clock_all - start_clock_all) / CLOCKS_PER_SEC);

  printf(
    "time total:%ld [s]\n",
    end_time_all - start_time_all);

  t_end = cputimeinsec();
  printf("%2.5f sec\n",t_end-t_start);
  char timefile[256];
  sprintf(timefile, "../crabGRP/time/calculation_time_u21339a4_ave500nsbyGPU.dat");
  FILE *fp_time;
  if ((fp_time = fopen(timefile, "wb")) == NULL){
         printf("file open error!!\n");
         exit(EXIT_FAILURE);
       }
  fprintf(fp_time, "in total %2.5f sec\ntime total:%ld [s]\noverall time:%2.5f sec\n",(double)(end_clock_all - start_clock_all) / CLOCKS_PER_SEC, end_time_all - start_time_all, t_end-t_start);
  fprintf(fp_time, "reading time : %d s\n", time_read);
  //fprintf(fp_time, "averaging + output time : %d s\n", time_aveout);
  fprintf(fp_time, "arranging bit time : %lf s\n", time_arranging_bit);
  fprintf(fp_time, "arranging bit time measured by cudaEvent: %e s\n", time_arrange_bit_bycudaE);
  fprintf(fp_time, "fread time : %lf s\n", time_fread);
  fprintf(fp_time, "averaging time : %d s\n", time_ave);
  fprintf(fp_time, "output time : %d s\n", time_out);
  fprintf(fp_time, "hipMemcpy time : %e\n", h_time_memcpy+h_time_memcpy_a);
  fprintf(fp_time, "FFT + mag time : %e\n", time_res[0]);
  fclose(fp_time);

  char paramfile[256];
  sprintf(paramfile, "u21339a4_ave500nsbyGPU.param");
  FILE *fp_param;
  if ((fp_param = fopen(paramfile, "wb")) == NULL){
         printf("file open error!!\n");
         exit(EXIT_FAILURE);
       }
  fprintf(fp_param, "Input data : %s\noutput data : %s\n",INPUT_RAWDATA, OUTPUT_DATA);
  fprintf(fp_param, "bitsize : %d\ntelescope : %s\nSR : %lld", INPUT_BITSIZE, observationdata,SR);
  //fprintf(fp_time, "averaging + output time : %d s\n", time_aveout);
  fprintf(fp_param, "freq width : %f\n", FREQ_DATA_WIDTH);
  fprintf(fp_param, "\n------------------------------------\n");
  fprintf(fp_param, "ch number(chnum) : %d\n ", (FFTP/2/64));
  fprintf(fp_param, "lowest freq : %f\nfreq width : %f\n",FREQ_DATA_LOW, FREQ_DATA_WIDTH);
  fprintf(fp_param, "dt(tsamp) : %e[s]\n ", (double)(FFTP)/SR);
  fprintf(fp_param, "dnu(foff): %f\n[MHz]", (double)(FREQ_DATA_WIDTH)/(FFTP/2/64));
  fclose(fp_param);
}





//--------end of this code
